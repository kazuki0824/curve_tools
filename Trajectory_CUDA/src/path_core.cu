#include "hip/hip_runtime.h"
/*
 * path_core.cu
 *
 *  Created on: 2018/10/31
 *      Author: kazuki
 */
#include "path_core.hpp"
std::vector<pSpline_polynomial>& getReference(float2 point[32] ,int pointcount, hipsparseHandle_t& cusparsehandle)
{

	pSpline_polynomial f[32];
	pSpline(point, f, pointcount, &cusparsehandle);
	std::vector<pSpline_polynomial> path;
	//TODO :個数は正しい?
	for (int i = 0; i < pointcount; ++i) {
		path.push_back(f[i]);
	}
}

float getMinimumTime(std::vector<pSpline_polynomial>& path, float maximum_velocity, float maximum_accel)
{
	float raw_time = 1.0 * path.size();

	float vtmp =0.0;
	float atmp = 0.0;

	for (int var = 0; var < path.size(); ++var) {
		for (float t = 0.0; t < 1.0; t += 0.05) {
			vtmp =fmax(length(path[var].speed(t)),vtmp);
			atmp =fmax(length(path[var].accel(t)),atmp);
		}
	}
    float scale = fmax( ( maximum_velocity / vtmp ) , ( maximum_accel / atmp ) );

    return raw_time * scale;
}


//float2.x=time; float2.y=angle
static std::vector<pSpline_polynomial> getReferenceAngle(float2 point[32], int pointcount, hipsparseHandle_t& cusparsehandle)
{
//TODO:
}


void DispatchTrajectory(float3 pose, float3 twist, std::vector<pSpline_polynomial>& path , float whole_time_for_reference)
{

}

