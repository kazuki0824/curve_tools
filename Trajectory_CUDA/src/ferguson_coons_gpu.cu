#include "hip/hip_runtime.h"
/*
 * ferguson_coons_gpu.cu
 *
 *  Created on: 2018/04/17
 *      Author: dvr1
 */


#include <stdio.h>
#include "hip/hip_vector_types.h"

#include "spline_gpu.hcu"

#include "ferguson_coons_gpu.hcu"


const float mat[16] =
{
		2, -2,  1,  1,
		-3,  3, -2, -1,
		0,  0,  1,  0,
		1,  0,  0,  0,
};

__host__ pSpline_polynomial Ferguson(hipblasHandle_t* handle, float2 v0, float2 v1, float2 x0, float2 x1)
{

	float _endpoint[8] = { x0.x, x1.x, v0.x, v1.x, x0.y, x1.y, v0.y, v1.y };

	float* points_gpu;
	float* matrix_gpu;

	float* output;

	hipMalloc(&output,sizeof(float) * 8);
	hipMalloc(&points_gpu,sizeof(float) * 8);
	hipMalloc(&matrix_gpu,sizeof(float) * 18);

	hipblasSetMatrix(4,2,sizeof(float),_endpoint, 4, points_gpu, 4);
	hipblasSetMatrix(4,4,sizeof(float),mat, 4, matrix_gpu, 4);

	
	float coef=1.0;
	float coef2 =0.0;

	hipblasStatus_t r = hipblasSgemm(*handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4,2,4,&coef,matrix_gpu,4,points_gpu,4,&coef2,output,4);

	float out_host[8];

	hipMemcpy(out_host, output, sizeof(float) * 8, hipMemcpyDeviceToHost);

	pSpline_polynomial x(out_host);

	hipFree(output);
	hipFree(points_gpu);
	hipFree(matrix_gpu);

	return x;
}
