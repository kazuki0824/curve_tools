#include "hip/hip_runtime.h"
/*
 * spline_gpu.cu
 *
 *  Created on: 2018/01/12
 *      Author: kazuki
 */

#include "hip/hip_vector_types.h"
#include "spline_gpu.hpp"
#include <stdio.h>

using namespace std;

//#define DEBUG

//http://www.yamamo10.jp/yamamoto/lecture/2006/5E/interpolation/interpolation_html/node3.html


//原点0,0
// pointcount は原点を含まない店の個数
__global__ void splineprep(float2 points[32], float temp_tdma_v[64], int pointcount =32) {

	float2 point = points[threadIdx.x];

	float2 point_prev = make_float2(__shfl_up(point.x,1),__shfl_up(point.y,1));

	if(threadIdx.x ==0) {
		point_prev.x = 0.0; point_prev.y = 0.0;
	}
	float2 dif = point - point_prev;

	float2 dif_next = make_float2(__shfl_down(dif.x,1), __shfl_down(dif.y,1));
	float2 difdif = dif_next - dif;

	// v_i/6
	temp_tdma_v[threadIdx.x] = difdif.x;
	temp_tdma_v[threadIdx.x+pointcount-1] = difdif.y;

}


__host__ __device__ float2 pSpline_polynomial::x_and_y(float in) {

	float t2=in*in;
	float t3=in*t2;
	return (this->a) * t3 + (this->b) * t2 + (this->c) * in + this->d;
}

__host__ __device__ float pSpline_polynomial::P2(float in)
{
	float t2=in*in;
	float x_ = ((this->a.x+this->a.x+this->a.x) * t2) + ((this->b.x+this->b.x) * in) + (this->c.x);
	float y_ = ((this->a.y+this->a.y+this->a.y) * t2) + ((this->b.y+this->b.y) * in) + (this->c.y);
	return ( x_ * x_ + y_ * y_ );
}

__host__ __device__ float pSpline_polynomial::curvature_radius(float in)
{

	float t2=in*in;
	float x_ = ((this->a.x+this->a.x+this->a.x) * t2) + ((this->b.x+this->b.x) * in) + (this->c.x);
	float x__ = (this->a.x * 6) * in + (this->b.x+this->b.x);
	float y_ = ((this->a.y+this->a.y+this->a.y) * t2) + ((this->b.y+this->b.y) * in) + (this->c.y);
	float y__ = (this->a.y * 6) * in + (this->b.y+this->b.y);

	float p2 ( x_ * x_ + y_ * y_ );
	float tmp = powf(p2,1.5);
	return tmp / (x_ * y__ - y_ * x__);
}
__host__ __device__ float2 pSpline_polynomial::accel(float in)
{
	float t2=in*in;
	float x__ = (this->a.x * 6) * in + (this->b.x+this->b.x);
	float y__ = (this->a.y * 6) * in + (this->b.y+this->b.y);
	float2 r = {X__,y__};
	return r;
}
__host__ __device__ float2 pSpline_polynomial::speed(float in)
{

	float t2=in*in;
	float x_ = ((this->a.x+this->a.x+this->a.x) * t2) + ((this->b.x+this->b.x) * in) + (this->c.x);
	float y_ = ((this->a.y+this->a.y+this->a.y) * t2) + ((this->b.y+this->b.y) * in) + (this->c.y);

	float2 r = {x_,y_};
	return r;
}

pSpline_polynomial::pSpline_polynomial(float args[8])
{
	this->a = { args[0] ,args[4] };
	this->b = { args[1] ,args[5] };
	this->c = { args[2] ,args[6] };
	this->d = { args[3] ,args[7] };
}



//static __device__ float out_x[16], out_y[16];


__global__ void spline(float2 points[], pSpline_polynomial f[32], int pointcount, float _tdma_u[64])
{
	float2 point = points[threadIdx.x];

	float2 point_prev = make_float2(__shfl_up(point.x,1),__shfl_up(point.y,1));

	__shared__ float u[2][33];
	u[0][threadIdx.x+1] = _tdma_u[threadIdx.x];
	u[1][threadIdx.x+1] = _tdma_u[threadIdx.x+pointcount-1]; //segfault注意

	if(threadIdx.x ==0) {
		point_prev.x = 0.0; point_prev.y = 0.0;
		u[0][0] = 0.0;
		u[0][pointcount] = 0.0;
		u[1][0] = 0.0;
		u[1][pointcount] = 0.0;
	}

	float2 dif = point - point_prev;


	pSpline_polynomial out;

	out.a.x = u[0][threadIdx.x + 1] -u[0][threadIdx.x];
	out.a.y = u[1][threadIdx.x + 1] -u[1][threadIdx.x];
	out.b.x = u[0][threadIdx.x] * 3.0;
	out.b.y = u[1][threadIdx.x] * 3.0;
	out.c.x = dif.x -(u[0][threadIdx.x]+u[0][threadIdx.x]+u[0][threadIdx.x+1]);
	out.c.y = dif.y -(u[1][threadIdx.x]+u[1][threadIdx.x]+u[1][threadIdx.x+1]);
	out.d = point_prev;
	out.start = point_prev;
	out.end = dif;

	const int integration_part = 2;

	float section[integration_part][3] = {0};
#pragma unroll
	for (int var = 0; var < integration_part; ++var) {
		section[var][0] = (float)(var) / (float)integration_part;
		section[var][2] = (float)(var+1) / (float)integration_part;
		section[var][1] = (section[var][0] + section[var][2]) /2.0;
	}
	/*
	 * 行列で工夫できたらいいと思った時があった
	__threadfence_block();
	*/
	out.length=0.0;
#pragma unroll
	for (int var = 0; var < integration_part; ++var) {
		out.length += (out.P(section[var][0])
						+ 4.0 * out.P(section[var][1])
							+ out.P(section[var][2]))
						* (section[var][2] - section[var][0]);
	}

	out.length /= 6.0;

	f[threadIdx.x] =out;

}

const float dl[32] = {0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
			1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
			1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
			1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,};
float du[32] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
				1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
				1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,
				1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 0.0,};
float d[32] = {4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0,
			4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0,
			4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0,
			4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0, 4.0, };

__host__ void pSpline(float2 points[], pSpline_polynomial f[32] , int pointcount, hipsparseHandle_t * handle)
{

	float2* pts;

	hipMalloc(&pts, sizeof(float2) * pointcount);

	hipMemcpy(pts,points,sizeof(float2) * pointcount,hipMemcpyHostToDevice);

	float* temp_tdma_v;

	hipMalloc(&temp_tdma_v , sizeof(float) * 64);
	#ifdef DEBUG
	printf("(%f,%f),",0.0,0.0);
	for (int var = 0; var < pointcount; ++var) {
		printf("(%f,%f),",points[var].x,points[var].y);
	}
	#endif


	splineprep<<<1,pointcount >>>(pts, temp_tdma_v,pointcount);

#ifdef DEBUG
	printf("\ninfo: Prep completed\n");
#endif

	pSpline_polynomial *p;
	hipMalloc((void**)&p, sizeof(pSpline_polynomial) * pointcount);

	float* dl_cuda;
	float* du_cuda;
	float* d_cuda;
	hipMalloc(&dl_cuda, sizeof(float) * (pointcount-1));
	hipMalloc(&du_cuda, sizeof(float) * (pointcount-1));
	hipMalloc(&d_cuda, sizeof(float) * (pointcount-1));
	hipMemcpy(dl_cuda, dl, sizeof(float) * (pointcount-1),hipMemcpyHostToDevice);
	hipMemcpy(du_cuda, du, sizeof(float) * (pointcount-1),hipMemcpyHostToDevice);
	hipMemcpy(d_cuda, d, sizeof(float) * (pointcount-1),hipMemcpyHostToDevice);


	hipDeviceSynchronize();



#ifdef DEBUG

	puts("temp_tdma_v");
	float matrix[64];
	hipMemcpy(matrix,temp_tdma_v,(pointcount-1)*2*sizeof(float), hipMemcpyDeviceToHost);
	for (int var = 0; var < (pointcount-1)*2; ++var) {
		printf("%f\n",matrix[var]);
	}

#endif


	//printf("%x %x %x %x\n",dl_cuda, d_cuda,du_cuda, temp_tdma_v);


    hipsparseStatus_t x = cusparseSgtsv(*handle, pointcount - 1, 2, dl_cuda, d_cuda, du_cuda, temp_tdma_v, pointcount - 1);

#ifdef DEBUG
	puts("temp_tdma_v(after gtsv)");
    hipMemcpy(matrix,temp_tdma_v,(pointcount-1)*2*sizeof(float), hipMemcpyDeviceToHost);
   	for (int var = 0; var < (pointcount-1)*2; ++var) {
   		printf("%f\n",matrix[var]);
   	}
#endif

    if (x!=0)
    	printf("err %d\n" , x);

	hipDeviceSynchronize();

	spline<<<1,pointcount>>>(pts, p, pointcount,temp_tdma_v);
	hipDeviceSynchronize();
	hipFree(temp_tdma_v);

	hipMemcpy(f,p,sizeof(pSpline_polynomial)*pointcount, hipMemcpyDeviceToHost);

	hipFree(p);
	hipFree(dl_cuda);
	hipFree(du_cuda);
	hipFree(d_cuda);

	hipFree(pts);
}

