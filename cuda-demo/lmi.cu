
#include <hip/hip_runtime.h>

#define SIZE 3
#define THREADX 3
#define THREADY 3
#define BLOCKX (SIZE/THREADX)
#define BLOCKY (SIZE/THREADY)
__global__ void matmulGPU(const float *A, const float *B, float *C) {
	int i,j,k;
	float
	sum=0.0f;
	int tx;
	__shared__
	float
	sB[THREADX];
	i	=
			blockIdx.x*blockDim.x+threadIdx.x;
	j	=
			blockIdx.y*blockDim.y+threadIdx.y;
	tx  =	threadIdx.x;
	for (k = 0; k < SIZE; k += THREADX) {
		sB[tx] = B[(k + tx) + SIZE * j];
		__syncthreads();
		for (int w = 0; w < THREADX; w++) {
			sum += A[i + SIZE * (k + w)] * sB[w];
		}
		__syncthreads();
	}
	C[i+SIZE*j] = sum;
}
