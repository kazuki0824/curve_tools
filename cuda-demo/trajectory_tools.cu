#include "hip/hip_runtime.h"
/*
 * trajectory_tools.cu
 *
 *  Created on: 2018/04/20
 *      Author: dvr1
 */

#include <vector>
#include <iostream>

#include "spline_gpu.hpp"

__host__ float max_speed_cpu(vector<pSpline_polynomial>& C, float step){
    int count = C.size();
    float v = 0.0;
    for (int var =0 ; var<count; var++)
    {
        for (float t = 0;t<1.0;t+=step)
        {
            v = fmaxf(v, length(C[var].speed(t)));
        }
    }
    return v;
}
__host__ float max_accel_cpu(vector<pSpline_polynomial>& C){
    int count = C.size();
    float v = 0.0;
    for (int var =0 ; var<count; var++)
    {
        for (float t = 0;t<1.0;t+=step)
        {
            a = fmaxf(a, length(C[var].accel(t)));
        }
    }
    return a;
}

__host__ __device__ static bool isInside(pSpline_polynomial* curve__part , float2 pos, float t, float v)
{
    const float boundary_radius = 0.1; //TODO: 定数
    boundary_radius *= v;
    return (boundary_radius > length(curve__part->x_and_y(t) - pos));
}

__host__ float sweep_cpu(vector<pSpline_polynomial>& C, float2 pos, float _speed, float step, float start = 0.0)
{
     int count = C.size();

    float end = (float)count;

    for (float t = start; t<end; t+=step)
    {
        int var = t / 1;
        if (isInside(&C[var], pos, t % 1.0, _speed))
            continue;
        else
            break;
    }
    return t;
}
