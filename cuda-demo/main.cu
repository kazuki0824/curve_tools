#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>


#include "hip/hip_vector_types.h"
#include "spline_gpu.hpp"
#include <vector>
#include <GL/freeglut.h>
#include "ferguson_coons_gpu.hcu"

hipblasHandle_t cublashandle;
hipsparseHandle_t cusparsehandle;
float2 v0={0.0,0.0};
void spl() {

	float2 point[6] = { { 0.1, 0.5 }, { 0.5, 0 }, { 0.5, 0.5 },{0.25,0.25} ,{-0.4,0.1}, {0.0,-0.5}};

	pSpline_polynomial f[32];
	pSpline(point, f, 6, &cusparsehandle);

	glBegin(GL_POINTS);
	//glBegin(GL_LINE_STRIP);

	for (int i = 0; i < 6; ++i) {
		pSpline_polynomial c = f[i];
		for (float t = 0.0; t < 1.0; t += 0.05) {
			float curve = fabsf(1.0/c.curvature_radius(t)) *0.3;
			glColor3f(curve, 0.7, 0.0);
			float2 p = c.x_and_y(t);
			glVertex2f(p.x+0.3, p.y+0.3);
			//printf("t=%f, (%f,%f)\n",t,p.x,p.y);
		}
	}
	glColor3f(1.0,0.0,0.0);
	glPointSize(0.5);

	float2 v1=f[0].speed(0.0); //v1 = length(v0) * normalize(v1); これはちゃんと動かない。最終的な速度はちゃんと渡そう

	float2 x0={0.0,0.0};
	float2 x1={0.3,0.3};

	pSpline_polynomial c = Ferguson(&cublashandle, v0, v1,x0, x1);

	for (float t = 0.0; t < 1.0; t += 0.05) {
		float2 p = c.x_and_y(t);
		glVertex2f(p.x, p.y);
	}

	glEnd();
	glColor3f(0.0,0.1,0.9);
	glBegin(GL_LINES);
	glVertex2f(0.0,0.0);
	glVertex2f(v0.x,v0.y);
	glEnd();


}

void display() {
	glClear(GL_COLOR_BUFFER_BIT);
	glBegin(GL_LINES);
	glColor3d(1.0, 1.0, 1.0);   // 基準線の描画
	glVertex2d(-1.0, 0.0);
	glVertex2d(1.0, 0.0);
	glVertex2d(0.0, 1.0);
	glVertex2d(0.0, -1.0);
	glEnd();

	spl();

	glFlush();

}

void init() {
	glClearColor(0.0, 0.0, 0.0, 1.0);   // 背景色　黒
	hipsparseCreate(&cusparsehandle);
	hipblasCreate(&cublashandle);

}
void deinit()
{
	hipblasDestroy(cublashandle);
	hipsparseDestroy(cusparsehandle);
}

void motion(int x , int y) {
	int get_ww = glutGet( GLUT_WINDOW_WIDTH );
    int get_wh = glutGet( GLUT_WINDOW_HEIGHT );
    float x_= (x - get_ww / 2) / (float)get_ww;
    float y_ = (get_wh / 2 -y) / (float)get_wh;

    v0={x_,y_};
    v0 *= 2.0;

	printf("X = %f : Y = %f\n" , v0.x , v0.y);

    glutPostRedisplay();
}

int main(int argc, char **argv) {

	glutInitWindowPosition(100, 200);   // ウィンドウ作成時の座標指定
	glutInitWindowSize(900, 900);        // ウィンドウサイズ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA); // Red Green Brue +α(透明度)= RGBA
	glutCreateWindow(argv[0]);         // ウィンドウを生成。 argv[0]だとファイル名がそのままウィンドウ名になる
	glutDisplayFunc(display);           // display関数内を表示
	init();
	glutMotionFunc(motion);

	glutMainLoop();

	return 0;
}
